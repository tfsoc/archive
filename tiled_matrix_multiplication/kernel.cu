/*
	C = A x B where
	A is m x n
	B is n x k
	C is m x k

	Assume the m,n,k are divisibles of BLOCK_SIZE
*/

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/device_functions.h>
#include <chrono>

// hide red underline for __syncthreads (nvidia forum based solution)
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#define BLOCK_SIZE 32

/*
	naive algorythm of matrix multiplication C = A * B
	where each thread calculates one element of C
*/
__global__ void matrixMultiplicationGPU1(float* a, float* b, float* c, int m, int n, int k) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	float sum = 0;
	if(!(col < k && row < m))return;

	//accumulate sum
	for (int i = 0; i < n; i++) {
		sum += a[row*n + i] * b[i*k + col];
	}

	//copy result to c
	c[row*k + col] = sum;
}

__global__ void matrixMultiplicationGPU(float* A, float* B, float* C, int N, int K) {
	//Block and Thread indices
	int bx = blockIdx.x, tx = threadIdx.x;
	int by = blockIdx.y, ty = threadIdx.y;


	//stores element of the block sub-matrix that is computed by the thread
	float Pvalue = 0;

	//loop over all sub-matrices of A and B required to compute the block sub-matrix
	for (int t = 0; t < N / BLOCK_SIZE; t++) {

		//shared memory declaration
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[ty][tx] = A[N * BLOCK_SIZE * by +  t * BLOCK_SIZE   + N * ty + tx];
		Bs[ty][tx] = B[BLOCK_SIZE * bx + t * K  *BLOCK_SIZE + K * ty + tx];

		__syncthreads();


#pragma unroll
		for (int i = 0; i < BLOCK_SIZE; i++) {
			Pvalue += As[ty][i] * Bs[i][tx];
		}
		
		__syncthreads();

	}

	C[K * BLOCK_SIZE * by + K * ty + BLOCK_SIZE * bx + tx] = Pvalue;
}


void matrixMultiplicationCPU(float* a, float* b, float* c, int m, int n, int k) {
	for (int i = 0; i < m; i++){
		for (int j = 0; j < k; j++){
			float res = 0;
			
			for (int ii = 0; ii < n; ii++) {
				res += a[i*n + ii] * b[ii*k + j];
			}
			
			c[i*k + j] = res;
		}
	}
}

bool equalMatrixCheckCPU(float* a, float* b, int m, int k) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < k; j++) {
			if (a[i*k + j] != b[i*k + j]) {
				printf("err: m1[%d][%d] != m2[%d][%d]\n", i, j, i, j);
				return false;
			}
		}
	}
	return true;
}

int main(){
	printf("Starting program...\n");

	int dId = 0;
	hipError_t err;
	
	hipSetDevice(dId);

	err = hipGetDevice(&dId);
	if (err != hipSuccess) {
		printf("%d: \t error %s (code %d) \n", __LINE__, hipGetErrorString(err), err); return 1;
	}

	dim3 dimsA(20 * BLOCK_SIZE, 2 * BLOCK_SIZE, 1);
	dim3 dimsB(40 * BLOCK_SIZE, 20 * BLOCK_SIZE, 1);
	dim3 dimsC(dimsB.x, dimsA.y, 1);

	if (dimsA.x != dimsB.y) {
		printf("product A x B does not exits! \n"); return 1;
	}

	
	int M = dimsA.y;
	int	N = dimsA.x; // == dimsB.y
	int K = dimsB.x;


	float *h_A, *h_B, *h_C, *h_C1;
	float *d_A, *d_B, *d_C;

	hipHostMalloc((void **)&h_A, sizeof(int)*M*N, hipHostMallocDefault);
	hipHostMalloc((void **)&h_B, sizeof(int)*N*K, hipHostMallocDefault);
	hipHostMalloc((void **)&h_C, sizeof(int)*M*K, hipHostMallocDefault);
	hipHostMalloc((void **)&h_C1, sizeof(int)*M*K, hipHostMallocDefault);
	
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_A[i*N + j] = (i + j) % 10;
		}
	}

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < K; j++) {
			h_B[i*K + j] = (i * j) % 10;
		}
	}

	hipMalloc((void **)&d_A, sizeof(float)*M*N);
	hipMalloc((void **)&d_B, sizeof(float)*N*K);
	hipMalloc((void **)&d_C, sizeof(float)*M*K);

	hipMemcpy(d_A, h_A, sizeof(float)*M*N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float)*N*K, hipMemcpyHostToDevice);



	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(dimsB.x / dimBlock.x, dimsA.y / dimBlock.y);
	//          (B.width / dimBlock.x, A.height/ dimBlock.y)
	

	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, NULL);
	matrixMultiplicationGPU<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N, K);
	hipEventRecord(stop, NULL);

	hipMemcpy(h_C, d_C, sizeof(float)*M*K, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&gpuTime, start, stop);
	printf("It took %f msec to perform %dx%d by %dx%d matrix multiplication on GPU.\n", gpuTime,M,N,N,K);

	hipDeviceSynchronize();
	


	auto begin = std::chrono::high_resolution_clock::now();
	matrixMultiplicationCPU(h_A, h_B, h_C1, M, N, K);
	auto end = std::chrono::high_resolution_clock::now();

	int cpuNano = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
	float cpuTime = cpuNano / 1000000.0;
	printf("It took %f msec to perform %dx%d by %dx%d matrix multiplication on CPU.\n", cpuTime, M, N, N, K);
	
	


	bool sameResult = equalMatrixCheckCPU(h_C, h_C1, M, K);
	if (sameResult)
		printf("Result is correct!\n");
	else
		printf("Check the code...\n");


	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Free memory...\n");
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipHostFree(h_C1);

	printf("Done...\n");




	return 0;
}


/*
	Reference:
		Slides on moodle
		"Shared Memory Matrix Multiplication" on https://www.youtube.com/watch?v=SqZaletdPCY
		CUDA by examples
		"Naive Matrix Multiplication CUDA" https://github.com/lzhengchun/matrix-cuda/blob/master/matrix_cuda.cu
		Forum topics from https://devtalk.nvidia.com/
*/